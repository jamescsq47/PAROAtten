#pragma once

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "../../kernels/residual.cuh"

at::Tensor add_residual(at::Tensor R, at::Tensor X){

  // X: [bs, 1, dim]
  int bs = X.size(0);
  int dim = X.size(2);

  at::Tensor O = torch::empty({bs, 1, dim}, 
        at::device(X.device()).dtype(at::ScalarType::Half));

  residual_kernel<<<dim3(DIV_UP(dim, 256), bs), dim3(128, 1)>>>(
            reinterpret_cast<half *>(X.data_ptr<at::Half>()),
            reinterpret_cast<half *>(R.data_ptr<at::Half>()),
            bs, dim,
            reinterpret_cast<half *>(O.data_ptr<at::Half>())
        );

  return O;
}
