#pragma once

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "../../kernels/rope.cuh"

void rope_permutation(at::Tensor Q, at::Tensor K, at::Tensor freq){

  // Q: [bs, H, W, dim]
  int bs = Q.size(0);
  int H = Q.size(1);
  int W = Q.size(2);
  int dim = Q.size(3);
  int hs = 128;

  rope_permutation_kernel<<<dim3(W, H, bs), dim3(128, 1)>>>(
            reinterpret_cast<half *>(Q.data_ptr<at::Half>()),
            reinterpret_cast<half *>(K.data_ptr<at::Half>()),
            freq.data_ptr<float>(), 
            bs, H, W, dim, hs
        );
}